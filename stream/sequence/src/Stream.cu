#include "Stream.cuh"

/**
 * @brief Sets up the chain that will be executed later.
 */
hipError_t Stream::initialize(
  const std::vector<char>& velopix_geometry,
  const std::vector<char>& ut_boards,
  const std::vector<char>& ut_geometry,
  const std::vector<char>& ut_magnet_tool,
  const std::vector<char>& scifi_geometry,
  const uint max_number_of_events,
  const bool param_do_check,
  const bool param_do_simplified_kalman_filter,
  const bool param_do_print_memory_manager,
  const bool param_run_on_x86,
  const std::string& param_folder_name_MC,
  const uint param_start_event_offset,
  const size_t reserve_mb,
  const uint param_stream_number,
  const Constants& param_constants
) {
  // Set stream and events
  cudaCheck(hipStreamCreate(&stream));
  cudaCheck(hipEventCreate(&cuda_generic_event));
  cudaCheck(hipEventCreate(&cuda_event_start));
  cudaCheck(hipEventCreate(&cuda_event_stop));

  // Set stream options
  stream_number = param_stream_number;
  do_check = param_do_check;
  do_simplified_kalman_filter = param_do_simplified_kalman_filter;
  do_print_memory_manager = param_do_print_memory_manager;
  run_on_x86 = param_run_on_x86;
  folder_name_MC = param_folder_name_MC;
  start_event_offset = param_start_event_offset;
  constants = param_constants;

  // Special case
  // Populate velo geometry
  cudaCheck(hipMalloc((void**)&dev_velo_geometry, velopix_geometry.size()));
  cudaCheck(hipMemcpyAsync(dev_velo_geometry, velopix_geometry.data(), velopix_geometry.size(), hipMemcpyHostToDevice, stream));

  // Populate UT boards and geometry
  cudaCheck(hipMalloc((void**)&dev_ut_boards, ut_boards.size()));
  cudaCheck(hipMemcpyAsync(dev_ut_boards, ut_boards.data(), ut_boards.size(), hipMemcpyHostToDevice, stream));

  cudaCheck(hipMalloc((void**)&dev_ut_geometry, ut_geometry.size()));
  cudaCheck(hipMemcpyAsync(dev_ut_geometry, ut_geometry.data(), ut_geometry.size(), hipMemcpyHostToDevice, stream));

  // Populate UT magnet tool values
  cudaCheck(hipMalloc((void**)&dev_ut_magnet_tool, ut_magnet_tool.size()));
  cudaCheck(hipMemcpyAsync(dev_ut_magnet_tool, ut_magnet_tool.data(), ut_magnet_tool.size(), hipMemcpyHostToDevice, stream));

  // Populate FT geometry
  cudaCheck(hipMalloc((void**)&dev_scifi_geometry, scifi_geometry.size()));
  cudaCheck(hipMemcpyAsync(dev_scifi_geometry, scifi_geometry.data(), scifi_geometry.size(), hipMemcpyHostToDevice, stream));

  // Memory allocations for host memory (copy back)
  cudaCheck(hipHostMalloc((void**)&host_velo_tracks_atomics, (2 * max_number_of_events + 1) * sizeof(int)));
  cudaCheck(hipHostMalloc((void**)&host_velo_track_hit_number, max_number_of_events * VeloTracking::max_tracks * sizeof(uint)));
  cudaCheck(hipHostMalloc((void**)&host_velo_track_hits, max_number_of_events * VeloTracking::max_tracks * VeloTracking::max_track_size * sizeof(Velo::Hit)));
  cudaCheck(hipHostMalloc((void**)&host_total_number_of_velo_clusters, sizeof(uint)));
  cudaCheck(hipHostMalloc((void**)&host_number_of_reconstructed_velo_tracks, sizeof(uint)));
  cudaCheck(hipHostMalloc((void**)&host_accumulated_number_of_hits_in_velo_tracks, sizeof(uint)));
  cudaCheck(hipHostMalloc((void**)&host_velo_states, max_number_of_events * VeloTracking::max_tracks * sizeof(Velo::State)));
  cudaCheck(hipHostMalloc((void**)&host_veloUT_tracks, max_number_of_events * VeloUTTracking::max_num_tracks * sizeof(VeloUTTracking::TrackUT)));
  cudaCheck(hipHostMalloc((void**)&host_atomics_veloUT, VeloUTTracking::num_atomics * max_number_of_events * sizeof(int)));
  cudaCheck(hipHostMalloc((void**)&host_accumulated_number_of_ut_hits, sizeof(uint)));
  cudaCheck(hipHostMalloc((void**)&host_accumulated_number_of_scifi_hits, sizeof(uint)));

  //Catboost initialization
  CatboostEvaluator evaluator("../../data/MuID-Run2-MC-570-v1.cb");
  model_float_feature_num = (int)evaluator.GetFloatFeatureCount();
  ObliviousTrees = evaluator.GetObliviousTrees();
  tree_num = ObliviousTrees->TreeSizes()->size();
  treeSplitsPtr_flat = ObliviousTrees->TreeSplits()->data();
  leafValuesPtr_flat = ObliviousTrees->LeafValues()->data();

  cudaCheck(hipHostMalloc((void***)&host_features, max_number_of_events * sizeof(float*)));
  cudaCheck(hipHostMalloc((void***)&host_borders, model_float_feature_num * sizeof(float*)));
  cudaCheck(hipHostMalloc((void**)&host_border_nums, model_float_feature_num * sizeof(int)));
  cudaCheck(hipHostMalloc((void***)&host_leaf_values, tree_num * sizeof(double*)));
  cudaCheck(hipHostMalloc((void***)&host_tree_splits, tree_num * sizeof(int*)));
  cudaCheck(hipHostMalloc((void**)&host_catboost_output, max_number_of_events * sizeof(float)));
  cudaCheck(hipHostMalloc((void**)&host_tree_sizes, tree_num * sizeof(int)));
  int index = 0;
  for (const auto& ff : *ObliviousTrees->FloatFeatures()) {
    int border_num = ff->Borders()->size();
    host_border_nums[index] = border_num;
    model_bin_feature_num += border_num;
    cudaCheck(hipMalloc((void**)&host_borders[index], border_num*sizeof(float)));
    cudaCheck(hipMemcpy(host_borders[index], ff->Borders()+1, border_num*sizeof(float),hipMemcpyHostToDevice));
    index++;
  }
  for (int i = 0; i < tree_num; i++) {
    host_tree_sizes[i] = ObliviousTrees->TreeSizes()->Get(i);
  }
  for (int i = 0; i < tree_num; i++) {
    int depth = host_tree_sizes[i];
    cudaCheck(hipMalloc((void**)&host_leaf_values[i], (1 << depth)*sizeof(double)));
    cudaCheck(hipMemcpy(host_leaf_values[i], leafValuesPtr_flat, (1 << depth)*sizeof(double), hipMemcpyHostToDevice));
    cudaCheck(hipMalloc((void**)&host_tree_splits[i], depth*sizeof(int)));
    cudaCheck(hipMemcpy(host_tree_splits[i], treeSplitsPtr_flat, depth*sizeof(int), hipMemcpyHostToDevice));
  
    leafValuesPtr_flat += (1 << depth);
    treeSplitsPtr_flat += depth;
  }

  // Define sequence of algorithms to execute
  sequence.set(sequence_algorithms());

  // Set options for each algorithm
  // (number of blocks, number of threads, stream, dynamic shared memory space)
  // Setup sequence items opts that are static and will not change
  // regardless of events on flight
  sequence.set_opts<seq::prefix_sum_single_block>(                      dim3(1), dim3(1024), stream);
  sequence.set_opts<seq::copy_and_prefix_sum_single_block>(             dim3(1), dim3(1024), stream);
  sequence.set_opts<seq::prefix_sum_single_block_velo_track_hit_number>(dim3(1), dim3(1024), stream);
  sequence.set_opts<seq::prefix_sum_single_block_ut_hits>(              dim3(1), dim3(1024), stream);
  sequence.set_opts<seq::prefix_sum_single_block_scifi_hits>(           dim3(1), dim3(1024), stream);

  // Get dependencies for each algorithm
  std::vector<std::vector<int>> sequence_dependencies = get_sequence_dependencies();

  // Get output arguments from the sequence
  std::vector<int> sequence_output_arguments = get_sequence_output_arguments();

  // Prepare dynamic scheduler
  scheduler = {get_sequence_names(), get_argument_names(),
    sequence_dependencies, sequence_output_arguments,
    reserve_mb * 1024 * 1024, do_print_memory_manager};

  // Malloc a configurable reserved memory
  cudaCheck(hipMalloc((void**)&dev_base_pointer, reserve_mb * 1024 * 1024));

  return hipSuccess;
}

void Stream::print_timing(
  const unsigned int number_of_events,
  const std::vector<std::pair<std::string, float>>& times
) {
  const auto total_time = times[times.size() - 1];
  std::string partial_times = "{\n";
  for (size_t i=0; i<times.size(); ++i) {
    if (i != times.size()-1) {
      partial_times += " " + times[i].first + "\t" + std::to_string(times[i].second) + "\t("
        + std::to_string(100 * (times[i].second / total_time.second)) + " %)\n";
    } else {
      partial_times += " " + times[i].first + "\t" + std::to_string(times[i].second) + "\t("
        + std::to_string(100 * (times[i].second / total_time.second)) + " %)\n}";
    }
  }

  info_cout << "stream #" << stream_number << ": "
    << number_of_events / total_time.second << " events/s"
    << ", partial timers (s): " << partial_times
    << std::endl;
}
