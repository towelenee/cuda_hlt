#include "SequenceSetup.cuh"

std::array<std::string, std::tuple_size<algorithm_tuple_t>::value> get_sequence_names() {
  std::array<std::string, std::tuple_size<algorithm_tuple_t>::value> a;
  a[seq::estimate_input_size] = "Estimate input size";
  a[seq::prefix_sum_reduce] = "Prefix sum reduce";
  a[seq::prefix_sum_single_block] = "Prefix sum single block";
  a[seq::prefix_sum_scan] = "Prefix sum scan";
  a[seq::masked_velo_clustering] = "Masked Velo clustering";
  a[seq::calculate_phi_and_sort] = "Calculate phi and sort";
  a[seq::fill_candidates] = "Fill candidates";
  a[seq::search_by_triplet] = "Search by triplet";
  a[seq::weak_tracks_adder] = "Weak tracks adder";
  a[seq::copy_and_prefix_sum_single_block] = "Copy and prefix sum single block";
  a[seq::copy_velo_track_hit_number] = "Copy Velo track hit number";
  a[seq::prefix_sum_reduce_velo_track_hit_number] = "Prefix sum reduce (2) Velo track hit number";
  a[seq::prefix_sum_single_block_velo_track_hit_number] = "Prefix sum single block (2) Velo track hit number";
  a[seq::prefix_sum_scan_velo_track_hit_number] = "Prefix sum scan (2) Velo track hit number";
  a[seq::consolidate_tracks] = "Consolidate tracks";
  a[seq::ut_calculate_number_of_hits] = "UT calculate number of hits";
  a[seq::prefix_sum_reduce_ut_hits] = "Prefix sum reduce (3) UT hits";
  a[seq::prefix_sum_single_block_ut_hits] = "Prefix sum single block (3) UT hits";
  a[seq::prefix_sum_scan_ut_hits] = "Prefix sum scan (3) UT hits";
  a[seq::decode_raw_banks] = "UT decode_raw_banks";
  a[seq::sort_by_y] = "Sort by Y";
  a[seq::veloUT] = "VeloUT tracking";
  a[seq::estimate_cluster_count] = "Estimate SciFi cluster count";
  a[seq::prefix_sum_reduce_ut_hits] = "Prefix sum reduce (4) SciFi hits";
  a[seq::prefix_sum_single_block_ut_hits] = "Prefix sum single block (4) SciFi hits";
  a[seq::prefix_sum_scan_ut_hits] = "Prefix sum scan (4) SciFi hits";
  a[seq::raw_bank_decoder] = "Decode SciFi raw banks";
  a[seq::scifi_sort_by_x] = "Sort SciFi hits by X";
  a[seq::gen_bin_features] = "Binary features generation for Catboost";
  a[seq::catboost_evaluator] = "Catboost model evaluation";

  return a;
}

std::array<std::string, std::tuple_size<argument_tuple_t>::value> get_argument_names() {
  std::array<std::string, std::tuple_size<argument_tuple_t>::value> a;
  a[arg::dev_raw_input] = "dev_raw_input";
  a[arg::dev_raw_input_offsets] = "dev_raw_input_offsets";
  a[arg::dev_estimated_input_size] = "dev_estimated_input_size";
  a[arg::dev_module_cluster_num] = "dev_module_cluster_num";
  a[arg::dev_module_candidate_num] = "dev_module_candidate_num";
  a[arg::dev_cluster_offset] = "dev_cluster_offset";
  a[arg::dev_cluster_candidates] = "dev_cluster_candidates";
  a[arg::dev_velo_cluster_container] = "dev_velo_cluster_container";
  a[arg::dev_tracks] = "dev_tracks";
  a[arg::dev_tracks_to_follow] = "dev_tracks_to_follow";
  a[arg::dev_hit_used] = "dev_hit_used";
  a[arg::dev_atomics_storage] = "dev_atomics_storage";
  a[arg::dev_tracklets] = "dev_tracklets";
  a[arg::dev_weak_tracks] = "dev_weak_tracks";
  a[arg::dev_h0_candidates] = "dev_h0_candidates";
  a[arg::dev_h2_candidates] = "dev_h2_candidates";
  a[arg::dev_rel_indices] = "dev_rel_indices";
  a[arg::dev_hit_permutation] = "dev_hit_permutation";
  a[arg::dev_velo_track_hit_number] = "dev_velo_track_hit_number";
  a[arg::dev_prefix_sum_auxiliary_array_2] = "dev_prefix_sum_auxiliary_array_2";
  a[arg::dev_velo_track_hits] = "dev_velo_track_hits";
  a[arg::dev_velo_states] = "dev_velo_states";
  a[arg::dev_ut_raw_input] = "dev_ut_raw_input";
  a[arg::dev_ut_raw_input_offsets] = "dev_ut_raw_input_offsets";
  a[arg::dev_ut_hit_offsets] = "dev_ut_hit_offsets";
  a[arg::dev_ut_hit_count] = "dev_ut_hit_count";
  a[arg::dev_ut_hits] = "dev_ut_hits";
  a[arg::dev_ut_hit_permutations] = "dev_ut_hit_permutations";
  a[arg::dev_veloUT_tracks] = "dev_veloUT_tracks";
  a[arg::dev_atomics_veloUT] = "dev_atomics_veloUT";
  a[arg::dev_scifi_raw_input] = "dev_scifi_raw_input";
  a[arg::dev_scifi_raw_input_offsets] = "dev_scifi_raw_input_offsets";
  a[arg::dev_scifi_hit_count] = "dev_scifi_hit_count";
  a[arg::dev_scifi_hit_permutations] = "dev_scifi_hit_permutations";
  a[arg::dev_prefix_sum_auxiliary_array_4] = "dev_prefix_sum_auxiliary_array_4";
  a[arg::dev_scifi_hits] = "dev_scifi_hits";
  a[arg::dev_borders] = "dev_borders";
  a[arg::dev_features] = "dev_features";
  a[arg::dev_border_nums] = "dev_border_nums";
  a[arg::dev_bin_features] = "dev_bin_features";
  a[arg::dev_tree_splits] = "dev_tree_splits";
  a[arg::dev_leaf_values] = "dev_leaf_values";
  a[arg::dev_tree_sizes] = "dev_tree_sizes";
  a[arg::dev_catboost_output] = "dev_catboost_output";
  return a;
}

std::vector<std::vector<int>> get_sequence_dependencies() {
  // Vector of dependecies for each algorithm
  std::vector<std::vector<int>> sequence_dependencies (
    std::tuple_size<argument_tuple_t>::value
  );

  sequence_dependencies[seq::estimate_input_size] = {
    arg::dev_raw_input,
    arg::dev_raw_input_offsets,
    arg::dev_estimated_input_size,
    arg::dev_module_cluster_num,
    arg::dev_module_candidate_num,
    arg::dev_cluster_candidates
  };
  sequence_dependencies[seq::prefix_sum_reduce] = {
    arg::dev_estimated_input_size,
    arg::dev_cluster_offset
  };
  sequence_dependencies[seq::prefix_sum_single_block] = {
    arg::dev_estimated_input_size,
    arg::dev_cluster_offset
  };
  sequence_dependencies[seq::prefix_sum_scan] = {
    arg::dev_estimated_input_size,
    arg::dev_cluster_offset
  };
  sequence_dependencies[seq::masked_velo_clustering] = {
    arg::dev_raw_input,
    arg::dev_raw_input_offsets,
    arg::dev_estimated_input_size,
    arg::dev_module_cluster_num,
    arg::dev_module_candidate_num,
    arg::dev_cluster_candidates,
    arg::dev_velo_cluster_container
  };
  sequence_dependencies[seq::calculate_phi_and_sort] = {
    arg::dev_estimated_input_size,
    arg::dev_module_cluster_num,
    arg::dev_velo_cluster_container,
    arg::dev_hit_permutation
  };
  sequence_dependencies[seq::fill_candidates] = {
    arg::dev_velo_cluster_container,
    arg::dev_estimated_input_size,
    arg::dev_module_cluster_num,
    arg::dev_h0_candidates,
    arg::dev_h2_candidates
  };
  sequence_dependencies[seq::search_by_triplet] = {
    arg::dev_velo_cluster_container,
    arg::dev_estimated_input_size,
    arg::dev_module_cluster_num,
    arg::dev_tracks,
    arg::dev_tracklets,
    arg::dev_tracks_to_follow,
    arg::dev_weak_tracks,
    arg::dev_hit_used,
    arg::dev_atomics_storage,
    arg::dev_h0_candidates,
    arg::dev_h2_candidates,
    arg::dev_rel_indices
  };
  sequence_dependencies[seq::weak_tracks_adder] = {
    arg::dev_velo_cluster_container,
    arg::dev_estimated_input_size,
    arg::dev_tracks,
    arg::dev_weak_tracks,
    arg::dev_hit_used,
    arg::dev_atomics_storage
  };
  sequence_dependencies[seq::copy_and_prefix_sum_single_block] = {
    arg::dev_atomics_storage
  };
  sequence_dependencies[seq::copy_velo_track_hit_number] = {
    arg::dev_tracks,
    arg::dev_atomics_storage,
    arg::dev_velo_track_hit_number
  };
  sequence_dependencies[seq::prefix_sum_reduce_velo_track_hit_number] = {
    arg::dev_velo_track_hit_number,
    arg::dev_prefix_sum_auxiliary_array_2
  };
  sequence_dependencies[seq::prefix_sum_single_block_velo_track_hit_number] = {
    arg::dev_velo_track_hit_number,
    arg::dev_prefix_sum_auxiliary_array_2
  };
  sequence_dependencies[seq::prefix_sum_scan_velo_track_hit_number] = {
    arg::dev_velo_track_hit_number,
    arg::dev_prefix_sum_auxiliary_array_2
  };
  sequence_dependencies[seq::consolidate_tracks] = {
    arg::dev_atomics_storage,
    arg::dev_tracks,
    arg::dev_velo_track_hit_number,
    arg::dev_velo_cluster_container,
    arg::dev_estimated_input_size,
    arg::dev_module_cluster_num,
    arg::dev_velo_track_hits,
    arg::dev_velo_states
  };
  sequence_dependencies[seq::ut_calculate_number_of_hits] = {
    arg::dev_ut_raw_input,
    arg::dev_ut_raw_input_offsets,
    arg::dev_ut_hit_offsets
  };
  sequence_dependencies[seq::prefix_sum_reduce_ut_hits] = {
    arg::dev_ut_hit_offsets,
    arg::dev_prefix_sum_auxiliary_array_3
  };
  sequence_dependencies[seq::prefix_sum_single_block_ut_hits] = {
    arg::dev_ut_hit_offsets,
    arg::dev_prefix_sum_auxiliary_array_3
  };
  sequence_dependencies[seq::prefix_sum_scan_ut_hits] = {
    arg::dev_ut_hit_offsets,
    arg::dev_prefix_sum_auxiliary_array_3
  };
  sequence_dependencies[seq::decode_raw_banks] = {
    arg::dev_ut_raw_input,
    arg::dev_ut_raw_input_offsets,
    arg::dev_ut_hits,
    arg::dev_ut_hit_offsets,
    arg::dev_ut_hit_count
  };
  sequence_dependencies[seq::sort_by_y] = {
    arg::dev_ut_hits,
    arg::dev_ut_hit_offsets,
    arg::dev_ut_hit_permutations
  };
  sequence_dependencies[seq::veloUT] = {
    arg::dev_ut_hits,
    arg::dev_ut_hit_offsets,
    arg::dev_atomics_storage,
    arg::dev_velo_track_hit_number,
    arg::dev_velo_track_hits,
    arg::dev_velo_states,
    arg::dev_veloUT_tracks,
    arg::dev_atomics_veloUT
  };
  sequence_dependencies[seq::estimate_cluster_count] = {
    arg::dev_scifi_raw_input,
    arg::dev_scifi_raw_input_offsets,
    arg::dev_scifi_hit_count
  };
  sequence_dependencies[seq::prefix_sum_reduce_scifi_hits] = {
    arg::dev_scifi_hit_count,
    arg::dev_prefix_sum_auxiliary_array_4
  };
  sequence_dependencies[seq::prefix_sum_single_block_scifi_hits] = {
    arg::dev_scifi_hit_count,
    arg::dev_prefix_sum_auxiliary_array_4
  };
  sequence_dependencies[seq::prefix_sum_scan_scifi_hits] = {
    arg::dev_scifi_hit_count,
    arg::dev_prefix_sum_auxiliary_array_4
  };
  sequence_dependencies[seq::raw_bank_decoder] = {
    arg::dev_scifi_raw_input,
    arg::dev_scifi_raw_input_offsets,
    arg::dev_scifi_hit_count,
    arg::dev_scifi_hits
  };
  sequence_dependencies[seq::scifi_sort_by_x] = {
    arg::dev_scifi_hits,
    arg::dev_scifi_hit_count,
    arg::dev_scifi_hit_permutations
  };
  sequence_dependencies[seq::gen_bin_features] = {
    arg::dev_borders,
    arg::dev_features,
    arg::dev_border_nums,
    arg::dev_bin_features
  };
  sequence_dependencies[seq::catboost_evaluator] = {
    arg::dev_tree_splits,
    arg::dev_leaf_values,
    arg::dev_tree_sizes,
    arg::dev_catboost_output,
    arg::dev_bin_features
  };

  return sequence_dependencies;
}

std::vector<int> get_sequence_output_arguments() {
  return {
    arg::dev_atomics_storage,
    arg::dev_velo_track_hit_number,
    arg::dev_velo_track_hits,
    arg::dev_atomics_veloUT,
    arg::dev_veloUT_tracks
  };
}
